
#include <hip/hip_runtime.h>
#include <omp.h>
#include <stdio.h>

#define chunk 20

void
mandel(int disp_width, int disp_height, int *array, int max_iter) {

  double 	scale_real, scale_imag;
  double 	x, y, u, v, u2, v2;
  int 	i, j, iter;

  #pragma omp parallel private(i, j, iter, x, y, u, v, u2, v2) \
    shared(scale_real, scale_imag, disp_width, disp_height, array, max_iter)
  {

  scale_real = 3.5 / (double)disp_width;
  scale_imag = 3.5 / (double)disp_height;

  #pragma omp for schedule(dynamic, chunk)
  for(i = 0; i < disp_width; i++) {

  	x = ((double)i * scale_real) - 2.25;

  	for(j = 0; j < disp_height; j++) {
  	    y = ((double)j * scale_imag) - 1.75;

  	    u    = 0.0;
  	    v    = 0.0;
  	    u2   = 0.0;
  	    v2   = 0.0;
  	    iter = 0;

  	    while ( u2 + v2 < 4.0 &&  iter < max_iter ) {
      		v = 2 * v * u + y;
      		u = u2 - v2 + x;
      		u2 = u*u;
      		v2 = v*v;
      		iter = iter + 1;
  	    }

  	    // if we exceed max_iter, reset to zero
  	    iter = iter == max_iter ? 0 : iter;

  	    array[i*disp_height + j] = iter;
  	}
  }

  } // end omp parallel
}
