
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define a(i,l) A[(i)*k + (l)]
#define b(l,j) B[(l)*n + (j)]
#define c(i,j) C[(i)*n + (j)]

// Declarations
extern "C" {
void matmult_gpu1(int m, int n, int k,double *h_A,double *h_B,double *h_C);
void matmult_gpu2(int m, int n, int k,double *h_A,double *h_B,double *h_C);
void matmult_gpu3(int m, int n, int k,double *h_A,double *h_B,double *h_C);
void matmult_gpu4(int m, int n, int k,double *h_A,double *h_B,double *h_C);
}

__global__ void matmult1(int m, int n, int k,double *A,double *B,double *C);
__global__ void matmult2(int m, int n, int k,double *A,double *B,double *C);
__global__ void matmult3(int m, int n, int k,double *A,double *B,double *C);
__global__ void matmult4(int m, int n, int k,double *A,double *B,double *C, int num_el);

void matmult_gpu1(int m, int n, int k,double *h_A,double *h_B,double *h_C) {

    double  *d_A, *d_B, *d_C;
    int size_A = m * k * sizeof(double);
    int size_B = k * n * sizeof(double);
    int size_C = m * n * sizeof(double);

    hipMalloc((void **)&d_A, size_A);
    hipMalloc((void **)&d_B, size_B);
    hipMalloc((void **)&d_C, size_C);

    dim3 dimBlock(1, 1, 1); // Num threads
    dim3 dimGrid(1, 1, 1); // Num blocks

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    hipMemset(d_C, 0, size_C);

    matmult1<<<dimGrid,dimBlock>>>(m, n, k, d_A, d_B, d_C);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}

__global__ void matmult1(int m, int n, int k,double *A,double *B,double *C) {

    int i,j,l;

    for (i = 0; i<m; i++) {
        for (l = 0; l<k; l++) {
            for (j = 0; j<n; j++) {
             	c(i,j) = c(i,j) + a(i,l) * b(l,j);
            }
        }
    }

}

void matmult_gpu2(int m, int n, int k,double *h_A,double *h_B,double *h_C) {

    double  *d_A, *d_B, *d_C;
    int size_A = m * k * sizeof(double);
    int size_B = k * n * sizeof(double);
    int size_C = m * n * sizeof(double);

    hipMalloc((void **)&d_A, size_A);
    hipMalloc((void **)&d_B, size_B);
    hipMalloc((void **)&d_C, size_C);

    dim3 dimBlock(16, 16, 1); // Num threads
    dim3 dimGrid(ceil((double)n/dimBlock.x), ceil((double)m/dimBlock.y), 1); // Num blocks

    //printf("x: %d, y: %d, z: %d\n", dimGrid.x, dimGrid.y, dimGrid.z);

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    hipMemset(d_C, 0, size_C);

    matmult2<<<dimGrid,dimBlock>>>(m, n, k, d_A, d_B, d_C);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}

__global__ void matmult2(int m, int n, int k,double *A,double *B,double *C) {

    int i,j,l;

    j = blockIdx.x * blockDim.x + threadIdx.x;
    i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < m && j < n) {
      for (l = 0; l<k; l++) {
         	c(i,j) = c(i,j) + a(i,l) * b(l,j);
      }
    }

}

void matmult_gpu3(int m, int n, int k,double *h_A,double *h_B,double *h_C) {

    double  *d_A, *d_B, *d_C;
    int size_A = m * k * sizeof(double);
    int size_B = k * n * sizeof(double);
    int size_C = m * n * sizeof(double);

    hipMalloc((void **)&d_A, size_A);
    hipMalloc((void **)&d_B, size_B);
    hipMalloc((void **)&d_C, size_C);

    dim3 dimBlock(16, 16, 1); // Num threads
    dim3 dimGrid((ceil((double)n/dimBlock.x)), ceil(((double)m/dimBlock.y) / 2), 1); // Num blocks

    //printf("x: %d, y: %d, z: %d\n", dimGrid.x, dimGrid.y, dimGrid.z);

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    hipMemset(d_C, 0, size_C);

    matmult3<<<dimGrid,dimBlock>>>(m, n, k, d_A, d_B, d_C);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}

__global__ void matmult3(int m, int n, int k,double *A,double *B,double *C) {

    int i,j,l;

    j = blockIdx.x * blockDim.x + threadIdx.x;
    i = (blockIdx.y * blockDim.y + threadIdx.y) * 2;

    if (i < m-1 && j < n) {
      for (l = 0; l<k; l++) {
         	c(i,j) = c(i,j) + a(i,l) * b(l,j);
            c(i+1,j) = c(i+1,j) + a(i+1,l) * b(l,j);
      }
    } else if (i == m-1 && j < n) {
        for (l = 0; l<k; l++) {
           	c(i,j) = c(i,j) + a(i,l) * b(l,j);
        }
    }

    /*
    j = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < m && j < n-1) {
      for (l = 0; l<k; l++) {
         	c(i,j) = c(i,j) + a(i,l) * b(l,j);
            c(i,j+1) = c(i,j+1) + a(i,l) * b(l,j+1);
      }
    } else if (i < m && j == n-1) {
        for (l = 0; l<k; l++) {
           	c(i,j) = c(i,j) + a(i,l) * b(l,j);
        }
    }
    */

}

void matmult_gpu4(int m, int n, int k,double *h_A,double *h_B,double *h_C) {

    double  *d_A, *d_B, *d_C;
    int size_A = m * k * sizeof(double);
    int size_B = k * n * sizeof(double);
    int size_C = m * n * sizeof(double);

    int num_el = 8;

    hipMalloc((void **)&d_A, size_A);
    hipMalloc((void **)&d_B, size_B);
    hipMalloc((void **)&d_C, size_C);

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16, 1); // Num threads
    dim3 dimGrid((ceil((double)n/dimBlock.x)), ceil(((double)m/dimBlock.y) / num_el), 1); // Num blocks

    hipMemset(d_C, 0, size_C);

    matmult4<<<dimGrid,dimBlock>>>(m, n, k, d_A, d_B, d_C, num_el);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}

__global__ void matmult4(int m, int n, int k,double *A,double *B,double *C, int num_el) {

    int i,j,l,s;

    j = blockIdx.x * blockDim.x + threadIdx.x;
    i = (blockIdx.y * blockDim.y + threadIdx.y) * num_el;

    if (i < m-num_el && j < n) {
      for (l = 0; l<k; l++) {
          c(i,j) = c(i,j) + a(i,l) * b(l,j);
          c(i+1,j) = c(i+1,j) + a(i+1,l) * b(l,j);
          c(i+2,j) = c(i+2,j) + a(i+2,l) * b(l,j);
          c(i+3,j) = c(i+3,j) + a(i+3,l) * b(l,j);
          c(i+4,j) = c(i+4,j) + a(i+4,l) * b(l,j);
          c(i+5,j) = c(i+5,j) + a(i+5,l) * b(l,j);
          c(i+6,j) = c(i+6,j) + a(i+6,l) * b(l,j);
          c(i+7,j) = c(i+7,j) + a(i+7,l) * b(l,j);
      }
    } else if (i >= m-num_el && j < n) {
        for (l = 0; l<k; l++) {
            for (s = i; s<m; s++) {
                c(s,j) = c(s,j) + a(s,l) * b(l,j);
            }
        }
    }

}
