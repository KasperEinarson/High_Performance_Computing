
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>

// Declarations
extern "C" {
void matmult_gpulib(int m, int n, int k,double *h_A,double *h_B,double *h_C);
}

void matmult_gpulib(int m,int n,int k,double *h_A,double *h_B,double *h_C) {

    double  *d_A, *d_B, *d_C;
    int size_A = m * k * sizeof(double);
    int size_B = k * n * sizeof(double);
    int size_C = m * n * sizeof(double);

    hipMalloc((void **)&d_A, size_A);
    hipMalloc((void **)&d_B, size_B);
    hipMalloc((void **)&d_C, size_C);

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    hipMemset(d_C, 0, size_C);

    hipblasHandle_t handle;
    double alpha = 1.0, beta = 0.0;
    int lda = k, ldb = n, ldc = n;

    hipblasCreate(&handle);

    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k,
      &alpha, d_A, lda, d_B, ldb, &beta, d_C, ldc);
    //cudaDeviceSynchronize();

    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    hipblasDestroy(handle);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
